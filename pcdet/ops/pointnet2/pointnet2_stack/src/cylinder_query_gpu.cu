#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
// #include <cstdlib>
// #include <ctime>
using namespace std;

#include "cylinder_query_gpu.h"
#include "cuda_utils.h"


__global__ void cylinder_query_kernel_stack(int B, int M, float radius, int nsample, \
    const float *new_xyz, const int *new_xyz_batch_cnt, const float *xyz, const int *xyz_batch_cnt, int *idx) {
    // :param xyz: (N1 + N2 ..., 3) xyz coordinates of the features
    // :param xyz_batch_cnt: (batch_size), [N1, N2, ...]
    // :param new_xyz: (M1 + M2 ..., 3) centers of the cylinder query
    // :param new_xyz_batch_cnt: (batch_size), [M1, M2, ...]
    // output:
    //      idx: (M, nsample)
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pt_idx >= M) return;

    int bs_idx = 0, pt_cnt = new_xyz_batch_cnt[0];
    for (int k = 1; k < B; k++){
        if (pt_idx < pt_cnt) break;
        pt_cnt += new_xyz_batch_cnt[k];
        bs_idx = k;
    }

    int xyz_batch_start_idx = 0;
    for (int k = 0; k < bs_idx; k++) xyz_batch_start_idx += xyz_batch_cnt[k];
    // for (int k = 0; k < bs_idx; k++) new_xyz_batch_start_idx += new_xyz_batch_cnt[k];

    new_xyz += pt_idx * 3;
    xyz += xyz_batch_start_idx * 3;
    idx += pt_idx * nsample;

    float radius2 = radius * radius;
    float new_x = new_xyz[0];
    float new_y = new_xyz[1];
    int n = xyz_batch_cnt[bs_idx];

    int cnt = 0;

    // int *A;
    // A = (int*)malloc(n*sizeof(int));
    // int tmp;
    // int tc;
    // for (int tc=0; tc<n; tc++)
    // {
    //     A[tc] = tc;
    // }
    // tmp = rand();
    // // srand((int)time(NULL));
    // // for (int tc=0; tc<n; tc++)
    // // {
    // //     tmp=rand();
    // //     // tmp = 0;
    // //     int val=A[tc];
    // //     A[tc]=A[tmp];
    // //     A[tmp]=val;
    // // }

    for (int kraw = 0; kraw < n; ++kraw) {
        // ktmp = rand()%10;
        // k = (kraw * ktmp) % n
        k = kraw;
        float x = xyz[k * 3 + 0];
        float y = xyz[k * 3 + 1];
        float d2 = (new_x - x) * (new_x - x) + (new_y - y) * (new_y - y);
        if (d2 < radius2){
            if (cnt == 0){
                for (int l = 0; l < nsample; ++l) {
                    idx[l] = k;
                }
            }
            idx[cnt] = k;
            ++cnt;
            if (cnt >= nsample) break;
        }
    }
    if (cnt == 0) idx[0] = -1;
}


void cylinder_query_kernel_launcher_stack(int B, int M, float radius, int nsample,
    const float *new_xyz, const int *new_xyz_batch_cnt, const float *xyz, const int *xyz_batch_cnt, int *idx){
    // :param xyz: (N1 + N2 ..., 3) xyz coordinates of the features
    // :param xyz_batch_cnt: (batch_size), [N1, N2, ...]
    // :param new_xyz: (M1 + M2 ..., 3) centers of the cylinder query
    // :param new_xyz_batch_cnt: (batch_size), [M1, M2, ...]
    // output:
    //      idx: (M, nsample)

    hipError_t err;

    dim3 blocks(DIVUP(M, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    cylinder_query_kernel_stack<<<blocks, threads>>>(B, M, radius, nsample, new_xyz, new_xyz_batch_cnt, xyz, xyz_batch_cnt, idx);
    // hipDeviceSynchronize();  // for using printf in kernel function
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}
